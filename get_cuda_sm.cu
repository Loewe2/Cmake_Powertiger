
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime_api.h>

int main(int argc, char *argv[])
{
    hipDeviceProp_t prop;
    hipError_t status;
    int device_count;
    int min_v = 0;

    status = hipGetDeviceCount(&device_count);
    if (status != hipSuccess) {
        fprintf(stderr,"cudaGetDeviceCount() failed: %s\n", hipGetErrorString(status));
        return -1;
    }
    for(int device_index=0; device_index < device_count; ++device_index){
        status = hipGetDeviceProperties(&prop, device_index);
        if (status != hipSuccess) {
            fprintf(stderr,"cudaGetDeviceProperties() for device %i failed: %s\n", device_index, hipGetErrorString(status));
            return -1;
        }
        int v = prop.major * 10 + prop.minor;
        if(min_v < v) min_v = v;
    }

   
    return min_v;
}